#include "hip/hip_runtime.h"
//
// Created by wulong on 7/15/18.
//

#include <iostream>
#include <vector>
#include <list>

#include "dpcuda.h"
#include "host_defines.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime_api.h"
#include "thrust/host_vector.h"

#include "../../../librarymsms/ICMzFile.h"
#include "../../../librarymsms/ICQuery.h"
#include "../../../librarymsms/ProteomicsDataTypes.h"
#include "../../../librarymsms/XMLFileParser.h"
#include "../../../librarymsms/ICGtInfoUpdate.h"
#include "../../../librarymsms/Util.h"

// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

// CUDA kernel to add elements of two arrays
__global__
void adduint(int n, uint16_t *x, uint16_t *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

__global__
void dp_norec(int specnum, uint16_t *all, int PeakNum, uint16_t *s, int query, int mzTopN, int tol) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    if (index == 0) {
//        cout << "stride: " << stride <<" block id="<<  blockIdx.x << " block size: " << blockDim.x << " gridDim/#blocks: " << gridDim.x << endl;
//        printf("stride: %d, blockid: %d, blocksize: %d, gridDim: %d\n", stride, blockIdx.x, blockDim.x, gridDim.x);
    }

    uint16_t *queryvector = all + PeakNum * query;

    s[index] = 0;
    for (int k = index; k < specnum; k += stride) {
        uint16_t *x = all + PeakNum * query, *y = all + index * PeakNum;

        // todo: the algorithm can be improved!
        for (int i = 0; i < mzTopN; i++) {
            if (x[i] == 0) break;
            for (int j = 0; j < mzTopN; j++) {
                if (y[j] == 0) { break; }

                if (x[i] > y[j]) // x[i] bigger
                {
                    if (x[i] - y[j] < tol) {
                        s[index] += (PeakNum - i) * (PeakNum - j);
                        break;
                    }
                } else {
                    if (y[j] - x[i] < tol) {

                        s[index] += (PeakNum - i) * (PeakNum - j);
                        break;
                    }
                }
            }

        }
    }


}

// todo: create another function to use GPU to calculate dot product

__global__
void dp_norec_vecform(int specnum, uint16_t *all, int PeakNum, uint16_t *s, int mzTopN, int tol,
                      uint16_t *vecform, bool debug=false, long debug_queryindex=-100) {
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    s[index] = 0;
    for (int k = index; k < specnum; k += stride) {
        uint16_t *y = all + index * PeakNum;
        unsigned long rec = 0;
        // todo: the algorithm can be improved!
        uint16_t used[51]={0};
        for (int i = 0; i < mzTopN; i++) {
            if(used[vecform[y[i]]]){}
            else
            {

                if(vecform[y[i]]) used[vecform[y[i]]] = 1;
                s[index] += vecform[y[i]] * (PeakNum - i) ;
#ifdef _DEBUG_INDEX_WITH_PRINT_
                if(debug_queryindex == index and debug and vecform[y[i]]>0)
                {
                    printf("%d -th mz: %d" ,i+1,y[i]);
                    printf("score : %d x %d -> %d\n", vecform[y[i]], PeakNum-i, s[index]);

                }
#endif


            }

        }
    }


}
#define _DEBUG_INDEX_WITH_PRINT_
// potential bug is here!
__global__
void dp_norec_vecform_on_index(int specnum, uint16_t *all, int PeakNum, uint16_t *s, int mzTopN,
                               int tol, uint16_t *vecform, long *gpuindexlist, int indexlistsize, bool debug=false, long debug_queryindex=-100) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int k = index; k < indexlistsize; k += stride) {
        s[k] = 0;
//        if(index==0) printf("topn=%d \nk=%d \nstride=%d\n", mzTopN, k,stride);
        uint16_t *y = all + gpuindexlist[k] * PeakNum;

        // todo: the algorithm can be improved!
        uint16_t used[51]={0};
        for (int i = 0; i < mzTopN; i++) {
            if(used[vecform[y[i]]]){
#ifdef _DEBUG_INDEX_WITH_PRINT_
               if(debug and debug_queryindex == gpuindexlist[k]) printf("used peak: %d\n", vecform[y[i]]);
#endif

            }
            else{
                if(vecform[y[i]]) used[vecform[y[i]]] = 1;
                s[k] += vecform[y[i]] * (PeakNum - i);

#ifdef _DEBUG_INDEX_WITH_PRINT_
                if(debug_queryindex == gpuindexlist[k] and debug and vecform[y[i]]>0)
                {
                    printf("%d -th mz: %d" ,i+1,y[i]);
                    printf("score : %d x %d -> %d\n", vecform[y[i]], PeakNum-i, s[k]);

                }
#endif
            }

        }
    }


}

__global__
void dp(int specnum, uint16_t *all, int PeakNum, uint16_t *s, int query, int mzTopN, int tol) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    s[index] = 0;
    for (int k = index; k < specnum; k += stride) {
        uint16_t *x = all + PeakNum * query, *y = all + index * PeakNum;
        uint64_t rec = 0;

        // todo: the algorithm can be improved!
        for (int i = 0; i < mzTopN; i++) {
            if (x[i] == 0) break;
            for (int j = 0; j < mzTopN; j++) {
                if (y[j] == 0) { break; }

                if (rec & (1L << j)) {
                    continue;
                }
                if (x[i] > y[j])  {
                    if (x[i] - y[j] < tol) {
                        s[index] += (PeakNum - i) * (PeakNum - j);
                        rec = (rec | (1L << j));
                        break;
                    }
                } else {
                    if (y[j] - x[i] < tol) {
                        s[index] += (PeakNum - i) * (PeakNum - j);
                        rec = (rec | (1L << j));
                        break;
                    }
                }
            }

        }
    }

}

// CUDA kernel to add elements of two arrays
//__global__
//void dpsorted(int specnum, uint16_t *all, int PeakNum, uint16_t *s, int query, int mzTopN, int tol)
//{
//    int index = blockIdx.x * blockDim.x + threadIdx.x;
//    int stride = blockDim.x * gridDim.x;
//
//
//    // preprocessing of y
////        vector<int> index(mzTopN);
////        iota(index.begin(), index.end(),0);
////        sort(index.begin(), index.end(), [](const int &a, const int &b){return y[a] < y[b];});
//
//
////        vector<int> recordj(mzTopN,0);
//    s[index] = 0;
//    for(int k = index; k < specnum; k += stride)
//    {
//        uint16_t *x = all + PeakNum * query, *y=all + index*PeakNum;
////        uint64_t rec = 0;
//
//
//
//        //---------------------
//        vector<int> yi(mzTopN), xi(mzTopN);
//        iota(yi.begin(), yi.end(), 0);
//        sort(yi.begin(), yi.end(), [y](const int &a, const int &b) { return y[a] < y[b]; });
//        iota(xi.begin(), xi.end(), 0);
//        sort(xi.begin(), xi.end(), [x](const int &a, const int &b) { return x[a] < x[b]; });
//        int i = 0, j = 0;
//
//        uint64_t recX = 0, recY = 0;
//
//
//        while (i < mzTopN and j < mzTopN) {
//            uint16_t alpha = x[xi[i]];
//            uint16_t beta = y[yi[j]];
//            if (alpha == 0) {
//                i++;
//                continue;
//            }
//            if (beta == 0) {
//                j++;
//                continue;
//            }
//            // we have minimal of x and minimal of y
//            if (alpha < beta) //x is smaller
//            {
//                if (beta - alpha <= tol) {
//                    // beta in alpha tolerance
//                    // Match!!
//                    // ------alpha-tol----------alpha-----------alpha+tol---------
//                    //----------------|-----------*-----^--------|-----------------
//                    //----------------------------------|<-beta-------------------
//                    // Keep looking for bigger one! ?? NO;  we are not sure whetherr alpha is the largest one!
//                    s[index] += (PeakNum - xi[i])*(PeakNum-yi[j]);//prescore[xi[i]][yi[j]];
//                    j++;
//                    i++;
////                    cout << score << " a=" << alpha << " b=" << beta << " xi="
////                        << xi[i] << " yi=" << yi[j] << " i="  << i <<" j=" << j << endl;
//                } else {
//                    i++; // beta is out of windows of [alpha +/- tol]
//                }
//            } else// beta<=alpha
//            {
//                if (alpha - beta <= tol) {
//                    // Beta is in windows, but less than alpha
//                    // ------alpha-tol----------alpha-----------alpha+tol---------
//                    //----------------|---^--------*--------------|-----------------
//                    //--------------------|<-beta-------------------
//                    s[index] += (PeakNum - xi[i])*(PeakNum-yi[j]);//prescore[xi[i]][yi[j]];
//                    j++;
//                    i++;
////                    cout << score << " a=" << alpha << " b=" << beta << " xi="
////                         << xi[i] << " yi=" << yi[j] << " i="  << i <<" j=" << j << endl;
//                } else {
//                    j++; // alpha is too big, beta need to catch up;
//                }
//            }
//        }
//
//
//    }
//
//
//
//}


// CUDA kernel to add elements of two arrays
__global__
void init_s(int n, uint16_t *s) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        s[i] = 0;
}

void run_cuda_dp_example() {
    // load data from file
    string mzfile = "/data/wulong/data/honeybee/all_mzXML.txt.mz";
    const int PeakNum = 50;
    long filebytes = 0;
    File::getfilesize(mzfile, filebytes);
    long peaknum = filebytes / 2;
    long specnum = peaknum / PeakNum;
    cout << "filebytes: " << filebytes << " peaknum " << peaknum << " specnum " << specnum << endl;

    // replace with new method
    uint16_t *all;
    uint16_t *s;
    hipMallocManaged(&all, peaknum * sizeof(uint16_t));
    hipMallocManaged(&s, specnum * sizeof(uint16_t));


    ifstream fin(mzfile.c_str(), ios::in | ios::binary);
    fin.read((char *) all, filebytes);
    fin.close();

    int mzTopN = PeakNum;
    int tol = 15;


    // copy data to gpu
    // let's move data to gpu after init
    // Prefetch the data to the GPU
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(all, peaknum * sizeof(uint16_t), device, NULL);
    hipMemPrefetchAsync(s, specnum * sizeof(uint16_t), device, NULL);
    int blockSize = 256;
    int numBlocks = (specnum + blockSize - 1) / blockSize;

    int query = 1312081;

    blockSize = 32;
    numBlocks = (specnum + blockSize - 1) / blockSize;
    for (int i = 1; i < 2; i++) {
        dp << < numBlocks, blockSize >> > (specnum, all, PeakNum, s, query, mzTopN, tol);
    }

    hipDeviceSynchronize();
    for (int i = 0; i < specnum and i < 1; i++) {
        cerr << s[i] << endl;
    }
    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < specnum; i++)
        maxError = fmax(maxError, fabs(s[i] - 0));
    std::cout << "Max error: " << maxError << std::endl;

    hipFree(all);
    hipFree(s);
}

CUDAScore::CUDAScore(string mzfile) : m_PeakNumPerSpec(50), m_filebytes(0), m_all(nullptr), m_s(nullptr),
                                      m_mzfile(mzfile) {
    cout << "Loading mz file to GPU: " << mzfile << endl;
    File::getfilesize(m_mzfile, m_filebytes);
    m_peaknum = m_filebytes / 2;
    m_specnum = m_peaknum / m_PeakNumPerSpec;

    hipMallocManaged(&m_all, m_peaknum * sizeof(uint16_t));
    cudaCheckErrors("Fail to allocate unified memory.");
    cout << "GPU RAM USED: MB: " << m_filebytes/1024/1024 << " #peak " << m_peaknum << " #spec " << m_specnum << endl;

    hipMallocManaged(&m_s, m_specnum * sizeof(uint16_t));
    cudaCheckErrors("Fail to allocate unified memory.");
    cout << "GPU RAM USED for store score of all: " << sizeof(uint16_t) * m_specnum/1024/1024 << "MB" << endl;
    gpuindexlist = nullptr;
    m_max_query_index_size = 500000;
    hipMallocManaged(&gpuindexlist, (m_max_query_index_size) * sizeof(long));
    cudaCheckErrors("Fail to allocate unified memory.");
    cout << "GPU RAM USED for index list: " << sizeof(long)*(m_max_query_index_size)/1024/1024 << "MB" << endl;

    int len = 1 + UINT16_MAX;
    cout << "Building vector of size  " << len << endl;
    hipMallocManaged(&vecform, (len) * sizeof(uint16_t));
    cudaCheckErrors("Fail to allocate unified memory.");

    ifstream fin(m_mzfile.c_str(), ios::in | ios::binary);
    fin.read((char *) m_all, m_filebytes);
    fin.close();

    int mzTopN = m_PeakNumPerSpec;
    int tol = 15;


    m_device = -1;

    hipGetDevice(&m_device);
    cudaCheckErrors("Fail to get CUDA device.");

    hipMemPrefetchAsync(m_all, m_peaknum * sizeof(uint16_t), m_device, NULL);
    cudaCheckErrors("Fetch cudaMem fails");
    hipMemPrefetchAsync(m_s, m_specnum * sizeof(uint16_t), m_device, NULL);
    cudaCheckErrors("Fetch cudaMem fails");
    cout << "--- GPU is ready! ----" << endl << endl;
}



CUDAScore::~CUDAScore() {
    cout << "Started to release GPU..." << endl;
    hipFree(m_all);
    hipFree(m_s);
    hipFree(vecform);
    hipFree(gpuindexlist);
    cout << "GPU released" << endl << endl;
}


void CUDAScore::Query(int topn, int tol, int queryindex, int blockSize) {

    cout << "Start query" << endl;
    int numBlocks = (m_specnum + blockSize - 1) / blockSize;

    dp_norec << < numBlocks, blockSize >> > (m_specnum, m_all, m_PeakNumPerSpec, m_s, queryindex, topn, tol);


    hipDeviceSynchronize();
    cout << "End of query" << endl;
}

void CUDAScore::scoreAllVecForm(int mzTopN, int tol, long queryindex, int blockSize, bool normalize) {
    int len = 1 + UINT16_MAX;
    uint16_t *queryspec = getSpecBy(queryindex);

    for (int i = 0; i < len; i++) {
        vecform[i] = 0;
    }
    for (int i = 0; i < m_PeakNumPerSpec; i++) {
        if (queryspec[i] == 0) break;
        int j = queryspec[i] >= tol ? queryspec[i] - tol : 0;
        int max_j = queryspec[i] >= len - tol ? len - tol : queryspec[i] + tol;
        while (j < max_j) {
            if (vecform[j] == 0 and j != 0) vecform[j] = (m_PeakNumPerSpec - i);
            j++;
        }
    }
    hipMemPrefetchAsync(vecform, len * sizeof(uint16_t), m_device, NULL);
    cudaCheckErrors("Fail in cudaMem");

    int numBlocks = (m_specnum + blockSize - 1) / blockSize;
    dp_norec_vecform << < numBlocks, blockSize >> >
                                         (m_specnum, m_all, m_PeakNumPerSpec, m_s, mzTopN, tol, vecform, true, 802792);

    hipDeviceSynchronize();

    if(normalize)
    {
        normalizationAllScore(queryindex);
    }
}


void CUDAScore::calcDotProduct(int TopNPeak, int tol, uint16_t *queryspec, int blockSize,
                               vector<long> &indexlist, vector<int> & scores){

	QueryfastOnIndexWithUINT16(TopNPeak, tol, queryspec, blockSize, indexlist);
	copyScore(indexlist, scores);
}
#include <mutex>
std::mutex dp_lock_gpu;
void CUDAScore::QueryfastOnIndexWithUINT16(int TopNPeak, int tol, uint16_t *queryspec, int blockSize,
                                           vector<long> &indexlist) {
    std::lock_guard<std::mutex> guard(dp_lock_gpu);
    int len = 1 + UINT16_MAX;

    toVectorForm(queryspec,tol,vecform, len);
    int indexlistsize = initGpuIndexList(indexlist);
    hipMemPrefetchAsync(vecform, len * sizeof(uint16_t), m_device, NULL);
    int numBlocks = (indexlistsize + blockSize - 1) / blockSize;
    long debug_index = 19266;
    debug_index = 658620;
    dp_norec_vecform_on_index << < numBlocks, blockSize >> >
    (m_specnum, m_all, m_PeakNumPerSpec, m_s, TopNPeak, tol, vecform, gpuindexlist, indexlistsize,false,debug_index);

    hipDeviceSynchronize();
}

void CUDAScore::queryFastOnIndex(int TopNPeak, int tol, long queryindex, int blockSize, vector<long> &indexlist) {
    uint16_t *queryspec = getSpecBy(queryindex);
    QueryfastOnIndexWithUINT16(TopNPeak, tol, queryspec, blockSize, indexlist);

}

uint16_t *CUDAScore::getscore() {
    return m_s;
}

long CUDAScore::getSpecNum() {
    return m_specnum;
}

int CUDAScore::getPeakNumPerSpec() const {
    return m_PeakNumPerSpec;
}

uint16_t *CUDAScore::getSpecBy(long queryindex) {
    return m_all + queryindex * getPeakNumPerSpec();
}

// todo: make this availabel on CPU also: Nov 16 2019
// Why we are still using this function?
vector<int> CUDAScore::distributionAll(int tol, long queryindex, bool normalize){
    int blockSize = 32;
    scoreAllVecForm(50, tol, queryindex, blockSize, normalize);
    const int MAX_TOP50_COS = 42925;
    const int MAX_SCORE = MAX_TOP50_COS + 1;
    vector<int> histogram(MAX_SCORE,0);
    for(long i = 0; i < m_specnum; i ++)    {
        int key = m_s[i];
        if(key < 0 or key > MAX_SCORE)        {
            cout << "Building histogram: invalid score: s[" << i <<  "]" << key << endl;
        } else{
            histogram[key] ++;
        }
    }
    return histogram;
}

void CUDAScore::normalizationAllScore(long queryindex) {
    const int MAX_SCORE=42925;
    double querynorm = getSquaredNorm(queryindex);

    for(long i = 0; i < m_specnum; i ++)    {
        double s = m_s[i];
        if(s>EPSILON)        {
            s/=sqrt(getSquaredNorm(i));
            s/=sqrt(querynorm);
        }

        s *= MAX_SCORE;
        if(s<0 or s>MAX_SCORE or i==802792)        {
            cout << "Normalize All scores: Invalid score s: " << s << " on index: " << i << endl;
            cout << "origin: " << m_s[i] << " norm: " << querynorm << " & " << getSquaredNorm(i) << endl;
        }
        m_s[i] = uint16_t(s)>MAX_SCORE? MAX_SCORE: uint16_t(s);
    }
}


int CUDAScore::initGpuIndexList(vector<long> &indexlist) {
    int indexlistsize = indexlist.size();
    if (indexlistsize >= m_max_query_index_size) {
        indexlistsize = m_max_query_index_size;
        cout << "[Warning] Index list exceed largest number allowed: " << m_max_query_index_size << ". Program will resize to max value" << endl;
    }
    for (int i = 0; i < m_max_query_index_size; i++) {
        if (i >= indexlistsize) {
            gpuindexlist[i] = -1;
        } else {

            gpuindexlist[i] = indexlist[i];  // todo: to be fixed long to int;
        }
    }
    hipMemPrefetchAsync(gpuindexlist, m_max_query_index_size * sizeof(long), m_device, NULL);  // wrong!!! fixed

    return indexlistsize;
}

void CUDAScore::toVectorForm(uint16_t *queryspec, int tol, uint16_t *vecform, int len) {
    const int PeakPerSpec = getPeakNumPerSpec();
    for (int i = 0; i < len; i++) {
        vecform[i] = 0;
    }
    for (int i = 0; i < PeakPerSpec; i++) {
        if (queryspec[i] == 0) break;
        int j = queryspec[i] >= tol ? queryspec[i] - tol : 0;
        int max_j = queryspec[i] >= len - tol ? len - tol : queryspec[i] + tol;
        while (j <= max_j) {  // fix this on GPU, Now CPU and GPU are the same
            if (vecform[j] == 0 and j != 0) vecform[j] = (PeakPerSpec - i);
            j++;
        }
    }
}

__global__
void pair_dp(long *index_pairs,uint16_t * query_spec_unimem, uint16_t * all, int index_pairs_len,int query_num,
        int PeakNum, int mzTopN, int tol, uint16_t *s, bool debug, long first_idx, long second_idx){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // todo: what is this?

    for (int k = index; k < index_pairs_len; k += stride) {
        s[k] = 0;
        long Idx = index_pairs[2*k], Idy = index_pairs[2*k+1];
        if(debug and Idx == first_idx and Idy == second_idx){
            printf("Score---\n");
        }
        uint16_t *y = query_spec_unimem + PeakNum * Idx, *x = all + Idy * PeakNum;
        uint64_t rec = 0;

        for (int i = 0; i < mzTopN; i++) {
            if (x[i] == 0) break;
            for (int j = 0; j < mzTopN; j++) {
                if (y[j] == 0) { break; }

                if (rec & (1L << j)) {
                    continue;
                }
                if (x[i] > y[j])
                {
                    if (x[i] - y[j] <= tol) {

                        s[k] += (PeakNum - i) * (PeakNum - j);

                        if(debug and Idx == first_idx and Idy == second_idx){
                            printf("%ld  %ld  %d <- %d x %d mz: %d vs %d\n",Idx, Idy, s[k],PeakNum-i,PeakNum-j,x[i],y[j]);
                        }

                        rec = (rec | (1L << j));

                        break;
                    }
                } else {
                    if (y[j] - x[i] <= tol) {

                        s[k] += (PeakNum - i) * (PeakNum - j);

                        if(debug and Idx == first_idx and Idy == second_idx){
                            printf("%ld  %ld  %d <- %d x %d mz: %d vs %d\n",Idx, Idy, s[k],PeakNum-i,PeakNum-j,x[i],y[j]);
                        }

                        rec = (rec | (1L << j));

                        break;
                    }
                }
            }

        }
    }
}

// Case 1: one query
// Case 2: many queries
void CUDAScore::dpscore(double tolerance, vector<vector<long>> &allRetIdx, int threadnum, vector<vector<float>> &accDist,
                   ICQuery &query,vector<vector<int>> &dpscores) {
    bool debug = false;
    long first = 5;
    long debug_index = 984171;

    // step 1. memory for queries
    uint16_t  *querySpecGpu;
    hipMallocManaged(&querySpecGpu, query.size()*getPeakNumPerSpec() * sizeof(uint16_t));
    long * index_pairs;
    int num = 0;
    for(auto x: allRetIdx) {
//        cout << "number of ret idx: " << x.size() << endl;
        num += x.size();
    }

    hipMallocManaged(&index_pairs, 2*num*sizeof(long));
    uint16_t *dps;
    hipMallocManaged(&dps,num*sizeof(uint16_t));
    //printf("allocated on GPU: querySpecGpu %ld; index_pairs: %ld; dps: %ld byts.\n",query.size()*getPeakNumPerSpec() * sizeof(uint16_t),2*num*sizeof(long),num*sizeof(uint16_t));

    // step 2. copy queryies to gpu
    for(int i = 0; i < query.size(); i ++)   {
        uint16_t  *p  = query.getPtrUint16(i);
        copy(p, p+getPeakNumPerSpec(),querySpecGpu + i*getPeakNumPerSpec() );
    }

    int k = 0;
    for(int i = 0; i < allRetIdx.size(); i ++)  {
        long first = i;
        for(int j = 0; j < allRetIdx[i].size();  j++)
        {
            long second = allRetIdx[i][j];
            index_pairs[2*k] = first;
            index_pairs[2*k+1] = second;
            k++;
        }
    }

    // step 3. calculate dp score for each pair of them
    int useTopN=50;
    int tol = 1;
    pair_dp<<<32,256>>>(index_pairs,querySpecGpu,m_all,num,query.size(),m_PeakNumPerSpec,useTopN,tol,dps, debug,first,debug_index);

    // step 4. copy score back
    hipDeviceSynchronize();

    // step 5. return
    hipFree(querySpecGpu);


    k = 0;
    for(int i = 0; i <query.size(); i ++)  {
        uint16_t  * queryspec = query.getPtrUint16(i);
        vector<long> &indexlist=allRetIdx[i];
        float querySquaredNorm = getSquaredNorm(queryspec);
        vector<float> accDist1st(indexlist.size(), 0);
        dpscores[i].assign(indexlist.size(),0);
        for (int j = 0; j < indexlist.size(); j++) {
            float candSquaredNorm = getSquaredNorm(indexlist[j]);
            accDist1st[j] = dps[k];
            dpscores[i][j] = dps[k];
            k++;
            if (candSquaredNorm > EPSILON and querySquaredNorm > EPSILON) {
                accDist1st[j] /= sqrt(candSquaredNorm * querySquaredNorm);
            }
            accDist1st[j] = sqrt(2.0) * sqrt(1.0 - accDist1st[j] < EPSILON ? 0 : 1.0 - accDist1st[j]);
        }
        accDist[i].swap(accDist1st);
    }

    hipFree(dps);
    hipFree(index_pairs);
}

void CUDAScore::copyScore(vector<long> &indexlist, vector<int> &scores) {
    int scoreLen = indexlist.size() > m_max_query_index_size? m_max_query_index_size: indexlist.size();
    scores.assign(scoreLen,0);
    for (int i = 0; i < scores.size(); i ++)    {
        scores[i] = getscore()[i];
    }
}

